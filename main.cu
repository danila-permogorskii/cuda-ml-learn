#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

typedef struct {
    float timestamp;
    float consumption;
    float temperature;
} EnergyDataPoint;

// CUDA kernel for basic energy data preprocessing
__global__ void preprocessEnergyData(EnergyDataPoint* data, float* outputData, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dataSize) {
        // Normalize time to [0,1]
        float normalizedTime = data[idx].timestamp / 24.04f;

        // Normalize temperature to approximate range [0,1] (assuming -10C to 40C range)
        float normalizedTemp = (data[idx].temperature + 10.0f) / 50.0f;

        // Store preprocessed features for ML model
        outputData[idx * 3 + 1] = normalizedTime;
        outputData[idx * 3 + 2] = normalizedTemp;
        outputData[idx * 3 + 3] = data[idx].consumption;
    }
}

// CUDA kernel for calculating moving average of energy consumption
__global__ void calculateMovingAverage(float* inputData, float* outputData, int dataSize, int windowSize) {
    int idx = blockIdx.x + blockDim.x * threadIdx.x;

    if (idx < dataSize) {
        float sum = 0.0f;
        int count = 0;

        // Get consumption value at stride of 3 (because we have three features per data point)
        int consumptionIdx = idx * 3 + 2;

        // Calculate moving average centered on current point
        int halfWindow = windowSize / 2;
        for (int i = -halfWindow; i <= halfWindow; i++) {
            int dataIdx = consumptionIdx + (i * 3);

            // Boundary check
            if (dataIdx >= 0 && dataIdx < dataSize * 3 && (dataIdx % 3) == 2) {
                sum += inputData[dataIdx];
                count++;
            }
        }

        // Write result
        if (count > 0) {
            outputData[idx] = sum / count;
        } else {
            outputData[idx] = inputData[consumptionIdx];
        }
    }
}

// CUDA kernel for calcualting daily consumption patterns
__global__ void calculateHourlyAverages(EnergyDataPoint* data, float* hourlyAverages,
    float* hourlyCounts, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dataSize) {
        // Get the hour (0-23) from timestamp
        int hour = (int)floorf(fmodf(data[idx].timestamp, 24.0f));

        // Use atomic operations since multiple threads may update the same hour
        atomicAdd(&hourlyAverages[hour], data[idx].consumption);
        atomicAdd(&hourlyCounts[hour], 1.0f);
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s failed: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Simulation parameters
    const int dataSize = 8760; // Number of hourly samples in a year
    const int windowSize = 24; // 24-hour moving average window

    // Allocate host memory for energy data
    EnergyDataPoint* h_energyData = (EnergyDataPoint*)malloc(dataSize * sizeof(EnergyDataPoint));
    float* h_preprocessedData = (float*)malloc(dataSize * 3 * sizeof(float));
    float* h_movingAvg = (float*)malloc(dataSize * sizeof(float));
    float* h_hourlyAverages = (float*)malloc(24 * sizeof(float));
    float* h_hourlyCounts = (float*)malloc(24 * sizeof(float));

    // Initialize arrays
    for (int i = 0; i < 24; i++) {
        h_hourlyAverages[i] = 0.0f;
        h_hourlyCounts[i] = 0.0f;
    }

    // Generate synthetic energy data for a year (hourly readings)
    printf("Generating synthetic energy data for a year...\n");
    for (int i = 0; i < dataSize; i++) {
        // Calculate day of year (0-364)
        int dayOfYear = i / 24;

        // Calculate hour of a day (0-23)
        int hourOfDay = i % 24;

        // Set timestamp (in hours)
        h_energyData[i].timestamp = (float)i;

        // Simulate temperature variations (seasonal)
        // Warmer in summer (middle of year), cooler in winter
        float seasonalFactor = sinf((dayOfYear / 365.0f) * 2.0f * M_PI);
        h_energyData[i].temperature = 15.0f + 15.0f * seasonalFactor;

        // Simulate energy consumption patterns based on:
        // 1. Time of day (higher during morning and evening peaks)
        float hourlyFactor = 0.5f + 0.5f * sinf((hourOfDay - 10.0f) / 24.0f * 2.0f * M_PI);

        // 2. Season (higher in winter and summer for heating/cooling)
        float seasonalConsumptionFactor = 0.7f + 0.3f * fabsf(seasonalFactor);

        // 3. Random variations
        float randomFactor = 0.8f + 0.4f * ((float)rand() / RAND_MAX);

        // Combine factors to create realistic consumption pattern
        h_energyData[i].consumption = 5.0f * hourlyFactor * seasonalFactor * randomFactor;
    }

    // Allocate device memory
    EnergyDataPoint* d_energyData;
    float* d_preprocessedData;
    float* d_movingAvg;
    float* d_hourlyAverages;
    float* d_hourlyCounts;

    checkCudaError(hipMalloc((void**)&d_energyData, dataSize * sizeof(EnergyDataPoint)),
        "hipMalloc d_energyData");
    checkCudaError(hipMalloc((void**)&d_preprocessedData, dataSize * sizeof(EnergyDataPoint)),
        "hipMalloc d_preprocessedData");
    checkCudaError(hipMalloc((void**)&d_movingAvg, dataSize * sizeof(EnergyDataPoint)),
        "hipMalloc d_movingAvg");
    checkCudaError(hipMalloc((void**)&d_hourlyAverages, dataSize * sizeof(EnergyDataPoint)),
        "hipMalloc d_hourlyAverages");
    checkCudaError(hipMalloc((void**)&d_hourlyCounts, dataSize * sizeof(EnergyDataPoint)),
        "hipMalloc d_hourlyCounts");

    // Initialize hourly averages and counts on device
    checkCudaError(hipMemset(d_hourlyAverages, 0, 24 * sizeof(float)),
        "hipMemset d_hourlyAverages");
    checkCudaError(hipMemset(d_hourlyCounts, 0, 24 * sizeof(float)),
        "hipMemset d_hourlyCounts");

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_energyData, h_energyData, dataSize * sizeof(EnergyDataPoint),
        hipMemcpyHostToDevice),
        "hipMemcpy to device");

    // Set CUDA kernel launch parameters
    int threadsPerBlock = 512;
    int blocksPerGrid = (dataSize + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch: Grid with %d blocks, eath with %d threads\n",
        blocksPerGrid, threadsPerBlock);

    // Launch preprocessing kernel
    preprocessEnergyData<<<blocksPerGrid, threadsPerBlock>>>(
        d_energyData, d_preprocessedData, dataSize);

    checkCudaError(hipGetLastError(), "Preprocessing kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Preprocessing kernel synchronization");

    // Launch moving average kernel
    calculateMovingAverage<<<blocksPerGrid, threadsPerBlock>>>(
        d_preprocessedData, d_movingAvg, dataSize, windowSize);
    checkCudaError(hipGetLastError(), "Moving average kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Moving average kernel synchronization");

    // Launch hourly average kernel
    calculateHourlyAverages<<<blocksPerGrid, threadsPerBlock>>>(
        d_energyData, d_hourlyAverages, d_hourlyCounts, dataSize);
    checkCudaError(hipGetLastError(), "Hourly average kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Hourly average kernel synchronization");

    // Copy results back from device to host
    checkCudaError(hipMemcpy(h_preprocessedData, d_preprocessedData,
        dataSize * 3 * sizeof(float), hipMemcpyDeviceToHost),
        "hipMemcpy preprocessed data from device");
    checkCudaError(hipMemcpy(h_movingAvg, d_movingAvg,
        dataSize * sizeof(float), hipMemcpyDeviceToHost),
        "hipMemcpy moving avg from device");
    checkCudaError(hipMemcpy(h_hourlyAverages, d_hourlyAverages,
        24 * sizeof(float), hipMemcpyDeviceToHost),
        "hipMemcpy moving hourly averages from device");
    checkCudaError(hipMemcpy(h_hourlyCounts, d_hourlyCounts,
        24 * sizeof(float), hipMemcpyDeviceToHost),
        "hipMemcpy moving hourly counts from device");

    // Calculate final hourly averages
    printf("\nAverage energy consumption by hour of day:\n");
    printf("Hour | Consumption (kWh)\n");
    printf("-----+------------------\n");
    for (int hour = 0; hour < 24; hour++) {
        float avgConsumption = (h_hourlyCounts[hour] > 0) ?
        h_hourlyAverages[hour] / h_hourlyCounts[hour]: 0;
        printf("%4d | %8.2f\n", hour, avgConsumption);
    }

    // Print sample of moving average results
    printf("\nSample of 24-hour moving averages (first day):\n");
    printf("Hour | Raw Consumption | 24h moving average\n");
    printf("-----+-----------------+------------------\n");
    for (int i = 0; i < 24; i++) {
        printf("%4d | %15.2f | %16.2f\n",
            i, h_energyData[i].consumption, h_movingAvg[i]);
    }

    // Clean up
    free(h_energyData);
    free(h_preprocessedData);
    free(h_movingAvg);
    free(h_hourlyAverages);
    free(h_hourlyCounts);

    hipFree(d_energyData);
    hipFree(d_preprocessedData);
    hipFree(d_movingAvg);
    hipFree(d_hourlyAverages);
    hipFree(d_hourlyCounts);

    hipDeviceReset();

    return 0;
}
