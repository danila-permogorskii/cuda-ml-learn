// Basic CUDA program demonstrating the structure for energy applications

#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel function
// This kernel will run on the GPU with many threads in parallel
__global__ void energyDataKernel(float *d_energyData, int dataSize) {
    // Calculate unique thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access beyond array bounds
    if (idx < dataSize) {
        // Simple computation: simulate processing energy consumption data
        // In a real application, this would be more complex
        d_energyData[idx] = d_energyData[idx] * 0.001f; // Convert to kwh if in Wh
    }
}


int main() {
    // Simulation: Number of energy consumption data points
    const int dataSize = 1024;
    size_t dataBytes = dataSize * sizeof(float);

    // Host (CPU) data arrays
    float *h_energyData = new float[dataSize];

    // Initialize with sample energy consumption data (in Wh)
    for (int i = 0; i < dataSize; i++) {
        h_energyData[i] = static_cast<float>(1000 + i % 1000); // Simulated Wh readings
    }

    // Device (GPU) data arrays
    float *d_energyData = nullptr;

    // Cuda kernel launch configuration
    int threadsPerBlock = 1024; // This is a specific value for my rtx 3050ti
    int blocksPerGrid = (dataSize + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory on the GPU
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void **) &d_energyData, dataBytes);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy data from host to device (CPU to GPU)
    cudaStatus = hipMemcpy(d_energyData, h_energyData, dataBytes, hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    printf("CUDA kernel launch: Grid with %d blocks, each with %d threads\n", blocksPerGrid, threadsPerBlock);

    // Launch CUDA kernel on GPU
    energyDataKernel<<<blocksPerGrid, threadsPerBlock>>>(d_energyData, dataSize);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "energyDataKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Wait for GPU to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy results from device to host (GPU to CPU)
    cudaStatus = hipMemcpy(h_energyData, d_energyData, dataBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy from device failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Print a few results to verify
    printf("Energy data Processing results (first 5 values, in kWh):\n");
    for (int i = 0; i < 5; i++) {
        printf("Data point %d: %.3f kWh\n", i, h_energyData[i]);
    }

    // Success!
    printf("CUDA processing successful!\n");

Error:
    // Free GPU memory
    hipFree(d_energyData);

    // Free CPU memory
    delete[] h_energyData;

    // Reset device
    hipDeviceReset();

    return (cudaStatus == hipSuccess) ? 0 : 1;
}
